
#include <hip/hip_runtime.h>

extern "C"
__global__ void mult(int *request,int *matrix,int *output)
{
	int index= threadIdx.x + blockIdx.x * blockDim.x;
	if(index<676*260)
	{
		//Durchiterieren tlength i d.h. auf 260 Blöcken
		//=> gridDim.x muss 260 sein
		int i=index / blockDim.x; //i =  index durch threadanzahl = wie oft schon durch
		int k=index-i*blockDim.x;
		//Durchiterieren qlength k mit jeweils 676 Threads
		if(matrix[k*260+i]==1&&request[k]==1) 
			output[i]++;
	}
	
	
}
